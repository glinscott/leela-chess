#include "hip/hip_runtime.h"
/*
  This file is part of Leela Chess Zero.
  Copyright (C) 2018 The LCZero Authors

  Leela Chess is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  Leela Chess is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with Leela Chess.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <cassert>
#include <functional>
#include <mutex>
#include "neural/factory.h"
#include "utils/bititer.h"
#include "utils/exception.h"

#include <hipblas.h>
#include <hipDNN.h>

namespace lczero {
namespace {

void cudnnError(hipdnnStatus_t status, const char *file, const int &line) {
  if (status != HIPDNN_STATUS_SUCCESS) {
    char message[128];
    sprintf(message, "CUDNN error: %s (%s:%d) ", hipdnnGetErrorString(status),
            file, line);
    throw Exception(message);
  }
}

const char* cublasGetErrorString(hipblasStatus_t status) {
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED: return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN: return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "unknown cublas error";
}

void cublasError(hipblasStatus_t status, const char *file, const int &line) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    char message[128];
    sprintf(message, "CUDNN error: %s (%s:%d) ", cublasGetErrorString(status),
        file, line);
    throw Exception(message);
  }
}

void hipError_t(hipError_t status, const char *file, const int &line) {
  if (status != hipSuccess) {
    char message[128];
    sprintf(message, "CUDA error: %s (%s:%d) ", hipGetErrorString(status),
        file, line);
    throw Exception(message);
  }
}


#define reportCUDNNErrors(status) cudnnError(status, __FILE__, __LINE__)
#define reportCUBLASErrors(status) cublasError(status, __FILE__, __LINE__)
#define reportCUDAErrors(status) hipError_t(status, __FILE__, __LINE__)

// 256 MB fixed scratch memory size (hardcoded for now)
static constexpr int kCudaScratchSize = 256 * 1024 * 1024;

// hard-coded for now, no point in going above this anyway (can possibly save
// memory by reducing this)
static constexpr int kMaxBatchSize = 1024;

// the Layer objects only hold memory for weights, biases, etc
// memory for input and output tensors is provided by caller of forwardEval

class BaseLayer {
 public:
  int GetC() const { return C; }
  int GetH() const { return H; }
  int GetW() const { return W; }

  BaseLayer(int c, int h, int w, BaseLayer *ip);
  size_t GetOutputSize(int N) const { return bpe_ * N * C * H * W; }

  // input2 is optional (skip connection)
  virtual void Eval(int N, float *output, const float *input,
                    const float *input2, float *scratch, hipdnnHandle_t cudnn,
                    hipblasHandle_t cublas) = 0;

 protected:
  static bool fp16_;
  static size_t bpe_;  // size of each element
  BaseLayer *input_;

  int C;  // output tensor dimensions
  int H;
  int W;
};

class ConvLayer : public BaseLayer {
 public:
  ConvLayer(BaseLayer *ip, int C, int H, int W, int size, int Cin,
            bool relu = false, bool bias = false);
  ~ConvLayer();
  void LoadWeights(float *pfilter, float *pBias = nullptr);
  void Eval(int N, float *output, const float *input, const float *input2,
            float *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  const int c_input_;
  const int filter_size_;
  const bool use_relu_;
  const bool use_bias_;

  float *biases = nullptr;
  float *weights = nullptr;

  hipdnnFilterDescriptor_t filter_desc_;
  hipdnnConvolutionDescriptor_t conv_desc_;
  hipdnnConvolutionFwdAlgo_t convAlgo;

  hipdnnTensorDescriptor_t bias_desc_;
  hipdnnTensorDescriptor_t in_tensor_desc_;
  hipdnnTensorDescriptor_t out_tensor_desc_;
  hipdnnActivationDescriptor_t activation_;
};

class SoftMaxLayer : public BaseLayer {
 public:
  SoftMaxLayer(BaseLayer *ip);
  void Eval(int N, float *output, const float *input, const float *input2,
            float *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  hipdnnTensorDescriptor_t out_tensor_desc_;
};

class BNLayer : public BaseLayer {
 public:
  BNLayer(BaseLayer *ip, bool relu);
  ~BNLayer();

  void LoadWeights(float *cpuMeans, float *cpuVar);
  void Eval(int N, float *output, const float *input, const float *input2,
            float *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  const bool use_relu_;
  float *means_ = nullptr;
  float *variances_ = nullptr;
};

class FCLayer : public BaseLayer {
 public:
  FCLayer(BaseLayer *ip, int C, int H, int W, bool relu, bool bias,
          bool tanh = false);
  ~FCLayer();

  void LoadWeights(float *cpuWeight, float *cpuBias);
  void Eval(int N, float *output, const float *input, const float *input2,
            float *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  const bool use_bias_;
  const bool use_relu_;
  const bool use_tanh_;
  float *weights_ = nullptr;
  float *biases_ = nullptr;
};

// Need memory for 3 data buffers
//  1. input for the layer
//  2. output of the layer
//  3. data from old layer for skip connection

/////////////////////////////////////////////////////////////////////////////
//                      Static variable Definations                        //
/////////////////////////////////////////////////////////////////////////////

// TODO: fp16 support
bool BaseLayer::fp16_ = false;
size_t BaseLayer::bpe_ = sizeof(float);

int divUp(int a, int b) { return (a + b - 1) / b; }

/////////////////////////////////////////////////////////////////////////////
//          Simple CUDA kernels used by certain layers                     //
/////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void addVectors_kernel(T *c, T *a, T *b, int size, int asize,
                                  int bsize, bool relu, bool useTanh) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    T aVal = 0;
    T bVal = 0;
    if (a) aVal = a[i % asize];
    if (b) bVal = b[i % bsize];

    T cVal = aVal + bVal;

    if (relu && (cVal < 0)) cVal = 0;

    if (useTanh) {
      // Ankan: actually it's sigmoid in leela-zero main branch??
      // see code in Network.cpp
      //    auto winrate_sig = (1.0f + std::tanh(winrate_out[0])) / 2.0f;
      // Different from lc0 branch? WHY ???
      // cVal = (1.0f + tanh(cVal)) / 2.0f;
      cVal = tanh(cVal);
    }

    c[i] = cVal;
  }
}

// adds two vectors (possibly of different sizes), also do optional relu
// activation_
template <typename T>
void addVectors(T *c, T *a, T *b, int size, int asize, int bsize, bool relu,
                bool useTanh) {
  const int blockSize = 256;
  int blocks = divUp(size, blockSize);

  addVectors_kernel<<<blocks, blockSize>>>(c, a, b, size, asize, bsize, relu,
                                           useTanh);
  reportCUDAErrors(hipGetLastError());
}

__global__ void batchNormForward_kernel(float *output, const float *input,
                                        const float *skipInput, int N, int C,
                                        int H, int W, const float *means,
                                        const float *varMultipliers,
                                        bool relu) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int wIndex = (index / (H * W)) % C;

  float el = input[index];
  float mean = means[wIndex];
  float varMulti = varMultipliers[wIndex];

  el -= mean;
  el *= varMulti;

  // TODO: figure out order of relu and skip connection
  if (skipInput) el += skipInput[index];

  if (relu && (el < 0)) el = 0;

  output[index] = el;
}

// works only on NCHW tensors
// each thread processes single element
void batchNormForward(float *output, const float *input, const float *skipInput,
                      int N, int C, int H, int W, float *means,
                      float *varMultipliers, bool relu) {
  int totalElements = N * C * H * W;
  const int blockSize = 256;
  int blocks = divUp(totalElements, blockSize);

  batchNormForward_kernel<<<blocks, blockSize>>>(
      output, input, skipInput, N, C, H, W, means, varMultipliers, relu);

  reportCUDAErrors(hipGetLastError());
}

BaseLayer::BaseLayer(int c, int h, int w, BaseLayer *ip)
    : C(c), H(h), W(w), input_(ip) {}

SoftMaxLayer::SoftMaxLayer(BaseLayer *ip)
    : BaseLayer(ip->GetC(), ip->GetH(), ip->GetW(), ip) {
  hipdnnCreateTensorDescriptor(&out_tensor_desc_);
}

void SoftMaxLayer::Eval(int N, float *output, const float *input,
                        const float *input2, float *scratch,
                        hipdnnHandle_t cudnn, hipblasHandle_t cublas) {
  float alpha = 1.0f, beta = 0.0f;

  // need to call this at Eval as 'N' changes :-/
  hipdnnSetTensor4dDescriptor(
      out_tensor_desc_, fp16_ ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, N, GetC(), GetH(), GetW());

  hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_ACCURATE,
                      HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha, out_tensor_desc_,
                      input, &beta, out_tensor_desc_, output);
}

ConvLayer::ConvLayer(BaseLayer *ip, int C, int H, int W, int filter, int Cin,
                     bool relu, bool bias)
    : BaseLayer(C, H, W, ip),
      filter_size_(filter),
      c_input_(Cin),
      use_relu_(relu),
      use_bias_(bias) {
  // allocate memory for weights (filter tensor) and biases
  size_t weightSize = bpe_ * Cin * C * filter_size_ * filter_size_;
  reportCUDAErrors(hipMalloc(&weights, weightSize));

  size_t biasSize = bpe_ * C;
  reportCUDAErrors(hipMalloc(&biases, biasSize));

  // create cudnn objects for various tensors, algorithms, etc
  hipdnnCreateFilterDescriptor(&filter_desc_);
  hipdnnCreateConvolutionDescriptor(&conv_desc_);
  hipdnnCreateTensorDescriptor(&out_tensor_desc_);
  hipdnnCreateTensorDescriptor(&in_tensor_desc_);
  hipdnnCreateTensorDescriptor(&bias_desc_);
  hipdnnCreateActivationDescriptor(&activation_);

  hipdnnSetFilter4dDescriptor(
      filter_desc_, fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT,
      fp16_ ? HIPDNN_TENSOR_NHWC
            : HIPDNN_TENSOR_NCHW,  // TODO: support fp16 evaluation
      GetC(), Cin, filter_size_, filter_size_);

  reportCUDNNErrors(hipdnnSetTensor4dDescriptor(
      bias_desc_, fp16_ ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, 1, C, 1, 1));

  int padding = filter_size_ / 2;
  const bool crossCorr = 1;

  hipdnnSetConvolution2dDescriptor(
      conv_desc_, padding, padding, 1, 1, 1, 1,
      crossCorr ? HIPDNN_CROSS_CORRELATION : HIPDNN_CONVOLUTION,
      fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT);

  // TODO: dynamic selection of algorithm!
  if (C > 32) {
    convAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
  } else {
    convAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  }

  if (use_relu_) {
    hipdnnSetActivationDescriptor(activation_, HIPDNN_ACTIVATION_RELU,
                                 HIPDNN_NOT_PROPAGATE_NAN, 0.0);
  } else {
    hipdnnSetActivationDescriptor(activation_, HIPDNN_ACTIVATION_PATHTRU,
                                 HIPDNN_NOT_PROPAGATE_NAN, 0.0);
  }
}

void ConvLayer::LoadWeights(float *pfilter, float *pBias) {
  size_t weightSize = bpe_ * c_input_ * C * filter_size_ * filter_size_;
  reportCUDAErrors(hipMemcpyAsync(weights, pfilter, weightSize, 
                                   hipMemcpyHostToDevice));

  size_t biasSize = bpe_ * C;
  if (pBias) {
    reportCUDAErrors(hipMemcpyAsync(biases, pBias, biasSize, 
                                     hipMemcpyHostToDevice));
  } else {
    reportCUDAErrors(hipMemset(biases, biasSize, 0));
  }
}

void ConvLayer::Eval(int N, float *output, const float *input,
                     const float *input2, float *scratch, hipdnnHandle_t cudnn,
                     hipblasHandle_t cublas) {
  reportCUDNNErrors(hipdnnSetTensor4dDescriptor(
      out_tensor_desc_, fp16_ ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, N, C, H, W));

  reportCUDNNErrors(hipdnnSetTensor4dDescriptor(
      in_tensor_desc_, fp16_ ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, N, c_input_, H, W));

  float alpha = 1.0f, beta = 0.0f;

  if (!(use_relu_ || use_bias_)) {
    reportCUDNNErrors(hipdnnConvolutionForward(
        cudnn, &alpha, in_tensor_desc_, input, filter_desc_, weights,
        conv_desc_, convAlgo, scratch, kCudaScratchSize, &beta,
        out_tensor_desc_, output));
  } else if (input2) {
    // fused bias + sum + relu!
    reportCUDNNErrors(cudnnConvolutionBiasActivationForward(
        cudnn, &alpha, in_tensor_desc_, input, filter_desc_, weights,
        conv_desc_, convAlgo, scratch, kCudaScratchSize, &alpha,
        out_tensor_desc_, input2, bias_desc_, biases, activation_,
        out_tensor_desc_, output));
  } else {
    reportCUDNNErrors(cudnnConvolutionBiasActivationForward(
        cudnn, &alpha, in_tensor_desc_, input, filter_desc_, weights,
        conv_desc_, convAlgo, scratch, kCudaScratchSize, &beta,
        out_tensor_desc_, output, bias_desc_, biases, activation_,
        out_tensor_desc_, output));
  }
}

ConvLayer::~ConvLayer() {
  reportCUDAErrors(hipFree(weights));
  reportCUDAErrors(hipFree(biases));
}

BNLayer::BNLayer(BaseLayer *ip, bool relu)
    : BaseLayer(ip->GetC(), ip->GetH(), ip->GetW(), ip), use_relu_(relu) {
  size_t weightSize = bpe_ * C;

  reportCUDAErrors(hipMalloc(&means_, weightSize));
  reportCUDAErrors(hipMalloc(&variances_, weightSize));
}

void BNLayer::LoadWeights(float *cpuMeans, float *cpuVar) {
  size_t weightSize = bpe_ * C;
  reportCUDAErrors(hipMemcpyAsync(means_, cpuMeans, weightSize, 
                                   hipMemcpyHostToDevice));
  reportCUDAErrors(hipMemcpyAsync(variances_, cpuVar, weightSize, 
                                   hipMemcpyHostToDevice));
}

void BNLayer::Eval(int N, float *output, const float *input,
                   const float *input2, float *scratch, hipdnnHandle_t cudnn,
                   hipblasHandle_t cublas) {
  batchNormForward(output, input, input2, N, C, H, W, means_, variances_,
                   use_relu_);
}

BNLayer::~BNLayer() {
  reportCUDAErrors(hipFree(means_));
  reportCUDAErrors(hipFree(variances_));
}

FCLayer::FCLayer(BaseLayer *ip, int C, int H, int W, bool relu, bool bias,
                 bool tanh)
    : BaseLayer(C, H, W, ip),
      use_relu_(relu),
      use_bias_(bias),
      use_tanh_(tanh) {
  size_t weightSize = bpe_ * C * H * W * ip->GetC() * ip->GetH() * ip->GetW();
  size_t biasSize = bpe_ * C * H * W;
  reportCUDAErrors(hipMalloc(&weights_, weightSize));
  if (use_bias_) {
    reportCUDAErrors(hipMalloc(&biases_, biasSize));
  } else {
    biases_ = nullptr;
  }
}

void FCLayer::LoadWeights(float *cpuWeight, float *cpuBias) {
  size_t weightSize =
      bpe_ * C * H * W * input_->GetC() * input_->GetH() * input_->GetW();

  reportCUDAErrors(hipMemcpyAsync(weights_, cpuWeight, weightSize, 
                                   hipMemcpyHostToDevice));
  if (use_bias_) {
    size_t biasSize = bpe_ * C * H * W;
    reportCUDAErrors(hipMemcpyAsync(biases_, cpuBias, biasSize, 
                                     hipMemcpyHostToDevice));
  }
}

void FCLayer::Eval(int N, float *outputTensor, const float *inputTensor,
                   const float *input2, float *scratch, hipdnnHandle_t cudnn,
                   hipblasHandle_t cublas) {
  float alpha = 1.0f, beta = 0.0f;
  int numOutputs = C * H * W;
  int numInputs = input_->GetC() * input_->GetH() * input_->GetW();

  if (fp16_) {
    // TODO: implement this!
    assert(0);
  } else {
    reportCUBLASErrors(hipblasSgemm(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, numOutputs, 
                                   N, numInputs, &alpha, weights_, numInputs, inputTensor, 
                                   numInputs, &beta, outputTensor, numOutputs));

    if (use_bias_ || use_relu_ || use_tanh_) {
      addVectors(outputTensor, biases_, outputTensor, numOutputs * N,
                 numOutputs, numOutputs * N, use_relu_, use_tanh_);
    }
  }
}

FCLayer::~FCLayer() {
  reportCUDAErrors(hipFree(weights_));
  reportCUDAErrors(hipFree(biases_));
}

class CudnnNetwork;

class CudnnNetworkComputation : public NetworkComputation {
 public:
  CudnnNetworkComputation(const CudnnNetwork *network) : network_(network) {}

  void AddInput(InputPlanes &&input) override {
    raw_input_.emplace_back(input);
  }

  void ComputeBlocking() override;

  int GetBatchSize() const override { return raw_input_.size(); }

  float GetQVal(int sample) const override { return out_val_[sample]; }
  float GetPVal(int sample, int move_id) const override {
    return out_pol_[sample][move_id];
  }

 private:
  // input
  std::vector<InputPlanes> raw_input_;

  static constexpr int kNumOutputPolicy = 1858;

  // output (TODO: try using hipHostAlloc to avoid the copy?)
  float out_pol_[kMaxBatchSize][kNumOutputPolicy];
  float out_val_[kMaxBatchSize];
  float input_planes_[kMaxBatchSize][kInputPlanes * 8 * 8];

  const CudnnNetwork *network_;
};

class CudnnNetwork : public Network {
 public:
  CudnnNetwork(Weights weights, const OptionsDict &options) {

    gpuId_ = options.GetOrDefault<int>("gpu", 0);

    int totalGPUs;
    reportCUDAErrors(hipGetDeviceCount(&totalGPUs));

    if (gpuId_ >= totalGPUs)
        throw Exception("Invalid GPU Id: " + std::to_string(gpuId_));

    // select GPU to run on (for *the current* thread)
    reportCUDAErrors(hipSetDevice(gpuId_));

    reportCUDNNErrors(hipdnnCreate(&cudnn_));
    reportCUBLASErrors(hipblasCreate(&cublas_));

    const int numInputPlanes = kInputPlanes;
    const int numFilters = weights.input.biases.size();

    numBlocks_ = weights.residual.size();

    // 0. process weights
    processConvBlock(weights.input, true);
    for (auto i = size_t{0}; i < numBlocks_; i++) {
      processConvBlock(weights.residual[i].conv1, true);
      processConvBlock(weights.residual[i].conv2, true);
    }
    processConvBlock(weights.policy);
    processConvBlock(weights.value);

    // 1. build the network, and copy the weights to GPU memory
    // input
    {
      auto inputConv = std::make_unique<ConvLayer>(nullptr, numFilters, 8, 8, 3,
                                                   numInputPlanes, true, true);
      inputConv->LoadWeights(&weights.input.weights[0],
                             &weights.input.biases[0]);
      network_.emplace_back(std::move(inputConv));
    }

    // residual block
    for (int block = 0; block < weights.residual.size(); block++) {
      auto conv1 = std::make_unique<ConvLayer>(getLastLayer(), numFilters, 8, 8,
                                               3, numFilters, true, true);
      conv1->LoadWeights(&weights.residual[block].conv1.weights[0],
                         &weights.residual[block].conv1.biases[0]);
      network_.emplace_back(std::move(conv1));

      auto conv2 = std::make_unique<ConvLayer>(getLastLayer(), numFilters, 8, 8,
                                               3, numFilters, true, true);
      conv2->LoadWeights(&weights.residual[block].conv2.weights[0],
                         &weights.residual[block].conv2.biases[0]);
      network_.emplace_back(std::move(conv2));
    }

    resi_last_ = getLastLayer();

    // policy head
    {
      auto convPol = std::make_unique<ConvLayer>(
          resi_last_, weights.policy.bn_means.size(), 8, 8, 1, numFilters);
      convPol->LoadWeights(&weights.policy.weights[0]);
      network_.emplace_back(std::move(convPol));

      auto BNPol = std::make_unique<BNLayer>(getLastLayer(), true);
      BNPol->LoadWeights(&weights.policy.bn_means[0],
                         &weights.policy.bn_stddivs[0]);
      network_.emplace_back(std::move(BNPol));

      auto FCPol = std::make_unique<FCLayer>(
          getLastLayer(), weights.ip_pol_b.size(), 1, 1, false, true);
      FCPol->LoadWeights(&weights.ip_pol_w[0], &weights.ip_pol_b[0]);
      network_.emplace_back(std::move(FCPol));

      auto softmaxPol = std::make_unique<SoftMaxLayer>(getLastLayer());
      network_.emplace_back(std::move(softmaxPol));
    }
    policy_out_ = getLastLayer();

    // Value head
    {
      auto convVal = std::make_unique<ConvLayer>(
          resi_last_, weights.value.bn_means.size(), 8, 8, 1, numFilters);
      convVal->LoadWeights(&weights.value.weights[0]);
      network_.emplace_back(std::move(convVal));

      auto BNVal = std::make_unique<BNLayer>(getLastLayer(), true);
      BNVal->LoadWeights(&weights.value.bn_means[0],
                         &weights.value.bn_stddivs[0]);
      network_.emplace_back(std::move(BNVal));

      auto FCVal1 = std::make_unique<FCLayer>(
          getLastLayer(), weights.ip1_val_b.size(), 1, 1, true, true);
      FCVal1->LoadWeights(&weights.ip1_val_w[0], &weights.ip1_val_b[0]);
      network_.emplace_back(std::move(FCVal1));

      auto FCVal2 =
          std::make_unique<FCLayer>(getLastLayer(), 1, 1, 1, false, true, true);
      FCVal2->LoadWeights(&weights.ip2_val_w[0], &weights.ip2_val_b[0]);
      network_.emplace_back(std::move(FCVal2));
    }
    value_out_ = getLastLayer();

    // 2. allocate GPU memory for running the network
    //    - three buffers of max size are enough (one to hold input, second to
    //    hold output and third to hold skip connection's input)
    size_t maxSize = resi_last_->GetOutputSize(kMaxBatchSize);
    for (auto &mem : tensor_mem_) {
      reportCUDAErrors(hipMalloc(&mem, maxSize));
      reportCUDAErrors(hipMemset(mem, 0, maxSize));
    }

    // printf("Allocated %d bytes of GPU memory to run the network\n", 3 *
    // maxSize);

    // 3. allocate scratch space (used internally by cudnn to run convolutions)
    reportCUDAErrors(hipMalloc(&scratch_mem_, kCudaScratchSize));
  }

  void forwardEval(const float *input, float *op_pol, float *op_val,
                   int batchSize) const {
    // printf(" ..%d.. ", batchSize);

    std::lock_guard<std::mutex> lock(lock_);

    // copy data from CPU memory to GPU memory
    reportCUDAErrors(hipMemcpyAsync(tensor_mem_[0], &input[0],
                     batchSize * kInputPlanes * network_[0]->GetH() *
                     network_[0]->GetW() * sizeof(float),
                     hipMemcpyHostToDevice));

    int l = 0;
    // input
    network_[l++]->Eval(batchSize, tensor_mem_[2], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // input conv

    // residual block
    for (int block = 0; block < numBlocks_; block++) {
      network_[l++]->Eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                          scratch_mem_, cudnn_, cublas_);  // conv1
      network_[l++]->Eval(batchSize, tensor_mem_[2], tensor_mem_[0],
                          tensor_mem_[2], scratch_mem_, cudnn_,
                          cublas_);  // conv2
    }

    // policy head
    network_[l++]->Eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // pol conv
    network_[l++]->Eval(batchSize, tensor_mem_[1], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // pol BN
    network_[l++]->Eval(batchSize, tensor_mem_[0], tensor_mem_[1], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // pol FC
    network_[l++]->Eval(batchSize, tensor_mem_[1], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_,
                        cublas_);  // pol softmax  // POLICY

    // value head
    network_[l++]->Eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // value conv
    network_[l++]->Eval(batchSize, tensor_mem_[2], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // value BN
    network_[l++]->Eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // value FC1
    network_[l++]->Eval(batchSize, tensor_mem_[2], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_,
                        cublas_);  // value FC2    // VALUE

    // copy results back to CPU memory
    reportCUDAErrors(hipMemcpyAsync(&op_pol[0], tensor_mem_[1],
                                     policy_out_->GetOutputSize(batchSize),
                                     hipMemcpyDeviceToHost));

    reportCUDAErrors(hipMemcpy(&op_val[0], tensor_mem_[2],
                                value_out_->GetOutputSize(batchSize),
                                hipMemcpyDeviceToHost));
  }

  ~CudnnNetwork() {
    for (auto mem : tensor_mem_) {
      if (mem) reportCUDAErrors(hipFree(mem));
    }
    if (scratch_mem_) reportCUDAErrors(hipFree(scratch_mem_));
    hipdnnDestroy(cudnn_);
    hipblasDestroy(cublas_);
  }

  std::unique_ptr<NetworkComputation> NewComputation() override {
    // set correct gpu id for this computation (as it might have been called from a different thread)
    reportCUDAErrors(hipSetDevice(gpuId_));
    return std::make_unique<CudnnNetworkComputation>(this);
  }

 private:
  hipdnnHandle_t cudnn_;
  hipblasHandle_t cublas_;
  int gpuId_;

  // currently only one NN Eval can happen a time (we can fix this if needed by
  // allocating more memory)
  mutable std::mutex lock_;

  int numBlocks_;
  std::vector<std::unique_ptr<BaseLayer>> network_;
  BaseLayer *getLastLayer() { return network_.back().get(); }

  BaseLayer *resi_last_;
  BaseLayer *policy_out_;
  BaseLayer *value_out_;

  float *tensor_mem_[3];
  float *scratch_mem_;

  void processConvBlock(Weights::ConvBlock &block, bool foldBNLayer = false) {
    const float epsilon = 1e-5f;

    // compute reciprocal of std-dev from the variances (so that it can be just
    // multiplied)
    std::vector<float> &stddev = block.bn_stddivs;
    for (auto &&w : stddev) {
      w = 1.0f / std::sqrt(w + epsilon);
    }

    // Biases are not calculated and are typically zero but some networks might
    // still have non-zero biases.
    // Move biases to batchnorm means to make the output match without having
    // to separately add the biases.
    for (auto j = size_t{0}; j < block.bn_means.size(); j++) {
      block.bn_means[j] -= block.biases[j];
      block.biases[j] = 0.0f;
    }

    // get rid of the BN layer by adjusting weights and biases of the
    // convolution idea proposed by Henrik Forst�n and first implemented in
    // leela go zero
    if (foldBNLayer) {
      const int outputs = block.biases.size();
      const int channels = block.weights.size() / (outputs * 3 * 3);

      for (auto o = 0; o < outputs; o++) {
        for (auto c = 0; c < channels; c++) {
          for (auto i = 0; i < 9; i++) {
            block.weights[o * channels * 9 + c * 9 + i] *= block.bn_stddivs[o];
          }
        }

        block.bn_means[o] *= block.bn_stddivs[o];
        block.bn_stddivs[o] = 1.0f;

        // Move means to convolution biases
        block.biases[o] = -block.bn_means[o];
        block.bn_means[o] = 0.0f;
      }
    }
  }
};

void CudnnNetworkComputation::ComputeBlocking() {
  // Convert raw_input to "expanded planes" - format the first convolutional
  // layer expects
  // TODO: can probably do this on the GPU if this becomes a bottleneck
  float *data = &(input_planes_[0][0]);
  memset(data, 0, sizeof(float) * GetBatchSize() * kInputPlanes * 8 * 8);
  auto iter = data;
  for (const auto &sample : raw_input_) {
    // CHECK_EQ(sample.size(), kInputPlanes);
    for (const auto &plane : sample) {
      for (auto bit : IterateBits(plane.mask)) {
        *(iter + bit) = plane.value;
      }
      iter += 64;
    }
  }

  network_->forwardEval(data, &(out_pol_[0][0]), &(out_val_[0]),
                        GetBatchSize());
}

}  // namespace

REGISTER_NETWORK("cudnn", CudnnNetwork, 110);

}  // namespace lczero
