#include "hip/hip_runtime.h"
/*
  This file is part of Leela Chess Zero.
  Copyright (C) 2018 The LCZero Authors

  Leela Chess is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  Leela Chess is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with Leela Chess.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <cassert>
#include <functional>
#include <mutex>
#include "neural/factory.h"
#include "utils/bititer.h"
#include "utils/exception.h"

#include <hipblas.h>
#include <hipDNN.h>

namespace lczero {
namespace {

void cudnnError(hipdnnStatus_t status, const char *file, const int &line) {
  if (status != HIPDNN_STATUS_SUCCESS) {
    char message[128];
    sprintf(message, "CUDNN error: %s (%s:%d) ", hipdnnGetErrorString(status),
            file, line);
    throw Exception(message);
  }
}

#define reportCUDNNErrors(status) cudnnError(status, __FILE__, __LINE__)

// 256 MB fixed scratch memory size (hardcoded for now)
static constexpr int kCudaScratchSize = 256 * 1024 * 1024;

// hard-coded for now, no point in going above this anyway (can possibly save
// memory by reducing this)
static constexpr int kMaxBatchSize = 1024;

// the Layer objects only hold memory for weights, biases, etc
// memory for input and output tensors is provided by caller of forwardEval

class BaseLayer {
 public:
  int getC() const { return C; }
  int getH() const { return H; }
  int getW() const { return W; }

  BaseLayer(int c, int h, int w, BaseLayer *ip);
  size_t getOutputSize(int N) const { return bpe * N * C * H * W; }

  // input2 is optional (skip connection)
  virtual void eval(int N, float *output, const float *input,
                    const float *input2, float *scratch, hipdnnHandle_t cudnn,
                    hipblasHandle_t cublas) = 0;

 protected:
  static bool fp16;
  static size_t bpe;  // size of each element
  int C, H, W;        // output tensor dimensions
  BaseLayer *input;
};

class ConvLayer : public BaseLayer {
 public:
  ConvLayer(BaseLayer *ip, int C, int H, int W, int size, int Cin,
            bool relu = false, bool bias = false);
  ~ConvLayer();
  void loadWeights(float *pfilter, float *pBias = nullptr);
  void eval(int N, float *output, const float *input, const float *input2,
            float *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  const int c_input_;
  const int filter_size_;
  const bool use_relu_;
  const bool use_bias_;

  float *biases = nullptr;
  float *weights = nullptr;

  hipdnnFilterDescriptor_t filter_desc_;
  hipdnnConvolutionDescriptor_t conv_desc_;
  hipdnnConvolutionFwdAlgo_t convAlgo;

  hipdnnTensorDescriptor_t bias_desc_;
  hipdnnTensorDescriptor_t in_tensor_desc_;
  hipdnnTensorDescriptor_t out_tensor_desc_;
  hipdnnActivationDescriptor_t activation_;
};

class SoftMaxLayer : public BaseLayer {
 public:
  SoftMaxLayer(BaseLayer *ip);
  void eval(int N, float *output, const float *input, const float *input2,
            float *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  hipdnnTensorDescriptor_t out_tensor_desc_;
};

class BNLayer : public BaseLayer {
 public:
  BNLayer(BaseLayer *ip, bool relu);
  ~BNLayer();

  void loadWeights(float *cpuMeans, float *cpuVar);
  void eval(int N, float *output, const float *input, const float *input2,
            float *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  const bool use_relu_;
  float *means_ = nullptr;
  float *variances_ = nullptr;
};

class FCLayer : public BaseLayer {
 public:
  FCLayer(BaseLayer *ip, int C, int H, int W, bool relu, bool bias,
          bool tanh = false);
  ~FCLayer();

  void loadWeights(float *cpuWeight, float *cpuBias);
  void eval(int N, float *output, const float *input, const float *input2,
            float *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  const bool use_bias_;
  const bool use_relu_;
  const bool use_tanh_;
  float *weights_ = nullptr;
  float *biases_ = nullptr;
};

// Each residual block has (4 kernels per block)
// A convolution of 128 filters of kernel size 3 × 3 with stride 1

// Batch normalisation
// A rectifier non - linearity

// A convolution of 128 filters of kernel size 3 × 3 with stride 1

// Batch normalisation
// A skip connection that adds the input to the block
// A rectifier non - linearity

// need implementations of
//  1. convolution layer (no bias/activation_ needed)
//  2. Fully connected layer (with optional bias, and optional relu),
//  3. batch normilization with optional sum (skip connection) and RELU

// Need memory for 3 data buffers
//  1. input for the layer
//  2. output of the layer
//  3. data from old layer for skip connection

/////////////////////////////////////////////////////////////////////////////
//                      Static variable Definations                        //
/////////////////////////////////////////////////////////////////////////////

// TODO: fp16 support
bool BaseLayer::fp16 = false;
size_t BaseLayer::bpe = sizeof(float);

int divUp(int a, int b) { return (a + b - 1) / b; }

/////////////////////////////////////////////////////////////////////////////
//          Simple CUDA kernels used by certain layers                     //
/////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void addVectors_kernel(T *c, T *a, T *b, int size, int asize,
                                  int bsize, bool relu, bool useTanh) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    T aVal = 0;
    T bVal = 0;
    if (a) aVal = a[i % asize];
    if (b) bVal = b[i % bsize];

    T cVal = aVal + bVal;

    if (relu && (cVal < 0)) cVal = 0;

    if (useTanh) {
      // Ankan: actually it's sigmoid in leela-zero main branch??
      // see code in Network.cpp
      //    auto winrate_sig = (1.0f + std::tanh(winrate_out[0])) / 2.0f;
      // Different from lc0 branch? WHY ???
      // cVal = (1.0f + tanh(cVal)) / 2.0f;
      cVal = tanh(cVal);
    }

    c[i] = cVal;
  }
}

// adds two vectors (possibly of different sizes), also do optional relu
// activation_
template <typename T>
void addVectors(T *c, T *a, T *b, int size, int asize, int bsize, bool relu,
                bool useTanh) {
  const int blockSize = 256;
  int blocks = divUp(size, blockSize);

  addVectors_kernel<<<blocks, blockSize>>>(c, a, b, size, asize, bsize, relu,
                                           useTanh);
}

__global__ void batchNormForward_kernel(float *output, const float *input,
                                        const float *skipInput, int N, int C,
                                        int H, int W, const float *means,
                                        const float *varMultipliers,
                                        bool relu) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int wIndex = (index / (H * W)) % C;

  float el = input[index];
  float mean = means[wIndex];
  float varMulti = varMultipliers[wIndex];

  el -= mean;
  el *= varMulti;

  // TODO: figure out order of relu and skip connection
  if (skipInput) el += skipInput[index];

  if (relu && (el < 0)) el = 0;

  output[index] = el;
}

// works only on NCHW tensors
// each thread processes single element
void batchNormForward(float *output, const float *input, const float *skipInput,
                      int N, int C, int H, int W, float *means,
                      float *varMultipliers, bool relu) {
  int totalElements = N * C * H * W;
  const int blockSize = 256;
  int blocks = divUp(totalElements, blockSize);

  batchNormForward_kernel<<<blocks, blockSize>>>(
      output, input, skipInput, N, C, H, W, means, varMultipliers, relu);
}

BaseLayer::BaseLayer(int c, int h, int w, BaseLayer *ip)
    : C(c), H(h), W(w), input(ip) {}

SoftMaxLayer::SoftMaxLayer(BaseLayer *ip)
    : BaseLayer(ip->getC(), ip->getH(), ip->getW(), ip) {
  hipdnnCreateTensorDescriptor(&out_tensor_desc_);
}

void SoftMaxLayer::eval(int N, float *output, const float *input,
                        const float *input2, float *scratch,
                        hipdnnHandle_t cudnn, hipblasHandle_t cublas) {
  float alpha = 1.0f, beta = 0.0f;

  // need to call this at eval as 'N' changes :-/
  hipdnnSetTensor4dDescriptor(
      out_tensor_desc_, fp16 ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16 ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, N, C, H, W);

  hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_ACCURATE,
                      HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha, out_tensor_desc_,
                      input, &beta, out_tensor_desc_, output);
}

ConvLayer::ConvLayer(BaseLayer *ip, int C, int H, int W, int filter, int Cin,
                     bool relu, bool bias)
    : BaseLayer(C, H, W, ip),
      filter_size_(filter),
      c_input_(Cin),
      use_relu_(relu),
      use_bias_(bias) {
  // allocate memory for weights (filter tensor) and biases
  size_t weightSize = bpe * Cin * C * filter_size_ * filter_size_;
  hipMalloc(&weights, weightSize);

  size_t biasSize = bpe * C;
  hipMalloc(&biases, biasSize);

  // create cudnn objects for various tensors, algorithms, etc
  hipdnnCreateFilterDescriptor(&filter_desc_);
  hipdnnCreateConvolutionDescriptor(&conv_desc_);
  hipdnnCreateTensorDescriptor(&out_tensor_desc_);
  hipdnnCreateTensorDescriptor(&in_tensor_desc_);
  hipdnnCreateTensorDescriptor(&bias_desc_);
  hipdnnCreateActivationDescriptor(&activation_);

  hipdnnSetFilter4dDescriptor(
      filter_desc_, fp16 ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT,
      fp16 ? HIPDNN_TENSOR_NHWC
           : HIPDNN_TENSOR_NCHW,  // TODO: support fp16 evaluation
      C, Cin, filter_size_, filter_size_);

  reportCUDNNErrors(hipdnnSetTensor4dDescriptor(
      bias_desc_, fp16 ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16 ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, 1, C, 1, 1));

  int padding = filter_size_ / 2;
  const bool crossCorr = 1;

  hipdnnSetConvolution2dDescriptor(
      conv_desc_, padding, padding, 1, 1, 1, 1,
      crossCorr ? HIPDNN_CROSS_CORRELATION : HIPDNN_CONVOLUTION,
      fp16 ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT);

  // TODO: dynamic selection of algorithm!
  if (C > 32) {
    convAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
  } else {
    convAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  }

  if (use_relu_) {
    hipdnnSetActivationDescriptor(activation_, HIPDNN_ACTIVATION_RELU,
                                 HIPDNN_NOT_PROPAGATE_NAN, 0.0);
  } else {
    hipdnnSetActivationDescriptor(activation_, HIPDNN_ACTIVATION_PATHTRU,
                                 HIPDNN_NOT_PROPAGATE_NAN, 0.0);
  }
}

void ConvLayer::loadWeights(float *pfilter, float *pBias) {
  size_t weightSize = bpe * c_input_ * C * filter_size_ * filter_size_;
  hipMemcpyAsync(weights, pfilter, weightSize, hipMemcpyHostToDevice);

  size_t biasSize = bpe * C;
  if (pBias) {
    hipMemcpyAsync(biases, pBias, biasSize, hipMemcpyHostToDevice);
  } else {
    hipMemset(biases, biasSize, 0);
  }
}

void ConvLayer::eval(int N, float *output, const float *input,
                     const float *input2, float *scratch, hipdnnHandle_t cudnn,
                     hipblasHandle_t cublas) {
  reportCUDNNErrors(hipdnnSetTensor4dDescriptor(
      out_tensor_desc_, fp16 ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16 ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, N, C, H, W));

  reportCUDNNErrors(hipdnnSetTensor4dDescriptor(
      in_tensor_desc_, fp16 ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16 ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, N, c_input_, H, W));

  float alpha = 1.0f, beta = 0.0f;

  if (!(use_relu_ || use_bias_)) {
    reportCUDNNErrors(hipdnnConvolutionForward(
        cudnn, &alpha, in_tensor_desc_, input, filter_desc_, weights,
        conv_desc_, convAlgo, scratch, kCudaScratchSize, &beta,
        out_tensor_desc_, output));
  } else if (input2) {
    // fused bias + sum + relu!
    reportCUDNNErrors(cudnnConvolutionBiasActivationForward(
        cudnn, &alpha, in_tensor_desc_, input, filter_desc_, weights,
        conv_desc_, convAlgo, scratch, kCudaScratchSize, &alpha,
        out_tensor_desc_, input2, bias_desc_, biases, activation_,
        out_tensor_desc_, output));
  } else {
    reportCUDNNErrors(cudnnConvolutionBiasActivationForward(
        cudnn, &alpha, in_tensor_desc_, input, filter_desc_, weights,
        conv_desc_, convAlgo, scratch, kCudaScratchSize, &beta,
        out_tensor_desc_, output, bias_desc_, biases, activation_,
        out_tensor_desc_, output));
  }
}

ConvLayer::~ConvLayer() {
  hipFree(weights);
  hipFree(biases);
}

BNLayer::BNLayer(BaseLayer *ip, bool relu)
    : BaseLayer(ip->getC(), ip->getH(), ip->getW(), ip), use_relu_(relu) {
  size_t weightSize = bpe * C;

  hipMalloc(&means_, weightSize);
  hipMalloc(&variances_, weightSize);
}

void BNLayer::loadWeights(float *cpuMeans, float *cpuVar) {
  size_t weightSize = bpe * C;
  hipMemcpyAsync(means_, cpuMeans, weightSize, hipMemcpyHostToDevice);
  hipMemcpyAsync(variances_, cpuVar, weightSize, hipMemcpyHostToDevice);
}

void BNLayer::eval(int N, float *output, const float *input,
                   const float *input2, float *scratch, hipdnnHandle_t cudnn,
                   hipblasHandle_t cublas) {
  batchNormForward(output, input, input2, N, C, H, W, means_, variances_,
                   use_relu_);
}

BNLayer::~BNLayer() {
  hipFree(means_);
  hipFree(variances_);
}

FCLayer::FCLayer(BaseLayer *ip, int C, int H, int W, bool relu, bool bias,
                 bool tanh)
    : BaseLayer(C, H, W, ip),
      use_relu_(relu),
      use_bias_(bias),
      use_tanh_(tanh) {
  size_t weightSize = bpe * C * H * W * ip->getC() * ip->getH() * ip->getW();
  size_t biasSize = bpe * C * H * W;
  hipMalloc(&weights_, weightSize);
  if (use_bias_) {
    hipMalloc(&biases_, biasSize);
  } else {
    biases_ = nullptr;
  }
}

void FCLayer::loadWeights(float *cpuWeight, float *cpuBias) {
  size_t weightSize =
      bpe * C * H * W * input->getC() * input->getH() * input->getW();
  hipMemcpyAsync(weights_, cpuWeight, weightSize, hipMemcpyHostToDevice);
  if (use_bias_) {
    size_t biasSize = bpe * C * H * W;
    hipMemcpyAsync(biases_, cpuBias, biasSize, hipMemcpyHostToDevice);
  }
}

void FCLayer::eval(int N, float *outputTensor, const float *inputTensor,
                   const float *input2, float *scratch, hipdnnHandle_t cudnn,
                   hipblasHandle_t cublas) {
  float alpha = 1.0f, beta = 0.0f;
  int numOutputs = C * H * W;
  int numInputs = input->getC() * input->getH() * input->getW();

  if (fp16) {
    // TODO: implement this!
    assert(0);
  } else {
    hipblasSgemm(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, numOutputs, N, numInputs,
                &alpha, weights_, numInputs, inputTensor, numInputs, &beta,
                outputTensor, numOutputs);

    if (use_bias_ || use_relu_ || use_tanh_) {
      addVectors(outputTensor, biases_, outputTensor, numOutputs * N,
                 numOutputs, numOutputs * N, use_relu_, use_tanh_);
    }
  }
}

FCLayer::~FCLayer() {
  hipFree(weights_);
  hipFree(biases_);
}

class CudnnNetwork;

class CudnnNetworkComputation : public NetworkComputation {
 public:
  CudnnNetworkComputation(const CudnnNetwork *network) : network_(network) {}

  void AddInput(InputPlanes &&input) override {
    raw_input_.emplace_back(input);
  }

  void ComputeBlocking() override;

  int GetBatchSize() const override { return raw_input_.size(); }

  float GetQVal(int sample) const override { return out_val_[sample]; }
  float GetPVal(int sample, int move_id) const override {
    return out_pol_[sample][move_id];
  }

 private:
  // input
  std::vector<InputPlanes> raw_input_;

  static constexpr int kNumOutputPolicy = 1858;

  // output (TODO: try using hipHostAlloc to avoid the copy?)
  float out_pol_[kMaxBatchSize][kNumOutputPolicy];
  float out_val_[kMaxBatchSize];
  float input_planes_[kMaxBatchSize][kInputPlanes * 8 * 8];

  const CudnnNetwork *network_;
};

class CudnnNetwork : public Network {
 public:
  CudnnNetwork(Weights weights) {
    // TODO: error checking!
    hipdnnCreate(&cudnn_);
    hipblasCreate(&cublas_);

    const int numInputPlanes = kInputPlanes;
    const int numFilters = weights.input.biases.size();

    numBlocks_ = weights.residual.size();

    // 0. process weights
    processConvBlock(weights.input, true);
    for (auto i = size_t{0}; i < numBlocks_; i++) {
      processConvBlock(weights.residual[i].conv1, true);
      processConvBlock(weights.residual[i].conv2, true);
    }
    processConvBlock(weights.policy);
    processConvBlock(weights.value);

    // 1. build the network, and copy the weights to GPU memory
    // input
    {
      auto inputConv = std::make_unique<ConvLayer>(nullptr, numFilters, 8, 8, 3,
                                                   numInputPlanes, true, true);
      inputConv->loadWeights(&weights.input.weights[0],
                             &weights.input.biases[0]);
      network_.emplace_back(std::move(inputConv));
    }

    // residual block
    for (int block = 0; block < weights.residual.size(); block++) {
      auto conv1 = std::make_unique<ConvLayer>(getLastLayer(), numFilters, 8, 8,
                                               3, numFilters, true, true);
      conv1->loadWeights(&weights.residual[block].conv1.weights[0],
                         &weights.residual[block].conv1.biases[0]);
      network_.emplace_back(std::move(conv1));

      auto conv2 = std::make_unique<ConvLayer>(getLastLayer(), numFilters, 8, 8,
                                               3, numFilters, true, true);
      conv2->loadWeights(&weights.residual[block].conv2.weights[0],
                         &weights.residual[block].conv2.biases[0]);
      network_.emplace_back(std::move(conv2));
    }

    resi_last_ = getLastLayer();

    // policy head
    {
      auto convPol = std::make_unique<ConvLayer>(
          resi_last_, weights.policy.bn_means.size(), 8, 8, 1, numFilters);
      convPol->loadWeights(&weights.policy.weights[0]);
      network_.emplace_back(std::move(convPol));

      auto BNPol = std::make_unique<BNLayer>(getLastLayer(), true);
      BNPol->loadWeights(&weights.policy.bn_means[0],
                         &weights.policy.bn_stddivs[0]);
      network_.emplace_back(std::move(BNPol));

      auto FCPol = std::make_unique<FCLayer>(
          getLastLayer(), weights.ip_pol_b.size(), 1, 1, false, true);
      FCPol->loadWeights(&weights.ip_pol_w[0], &weights.ip_pol_b[0]);
      network_.emplace_back(std::move(FCPol));

      auto softmaxPol = std::make_unique<SoftMaxLayer>(getLastLayer());
      network_.emplace_back(std::move(softmaxPol));
    }
    policy_out_ = getLastLayer();

    // Value head
    {
      auto convVal = std::make_unique<ConvLayer>(
          resi_last_, weights.value.bn_means.size(), 8, 8, 1, numFilters);
      convVal->loadWeights(&weights.value.weights[0]);
      network_.emplace_back(std::move(convVal));

      auto BNVal = std::make_unique<BNLayer>(getLastLayer(), true);
      BNVal->loadWeights(&weights.value.bn_means[0],
                         &weights.value.bn_stddivs[0]);
      network_.emplace_back(std::move(BNVal));

      auto FCVal1 = std::make_unique<FCLayer>(
          getLastLayer(), weights.ip1_val_b.size(), 1, 1, true, true);
      FCVal1->loadWeights(&weights.ip1_val_w[0], &weights.ip1_val_b[0]);
      network_.emplace_back(std::move(FCVal1));

      auto FCVal2 =
          std::make_unique<FCLayer>(getLastLayer(), 1, 1, 1, false, true, true);
      FCVal2->loadWeights(&weights.ip2_val_w[0], &weights.ip2_val_b[0]);
      network_.emplace_back(std::move(FCVal2));
    }
    value_out_ = getLastLayer();

    // 2. allocate GPU memory for running the network
    //    - three buffers of max size are enough (one to hold input, second to
    //    hold output and third to hold skip connection's input)
    size_t maxSize = resi_last_->getOutputSize(kMaxBatchSize);
    for (auto &mem : tensor_mem_) {
      hipMalloc(&mem, maxSize);
      hipMemset(mem, 0, maxSize);
    }

    // printf("Allocated %d bytes of GPU memory to run the network\n", 3 *
    // maxSize);

    // 3. allocate scratch space (used internally by cudnn to run convolutions)
    hipMalloc(&scratch_mem_, kCudaScratchSize);
  }

  void forwardEval(const float *input, float *op_pol, float *op_val,
                   int batchSize) const {
    // printf(" ..%d.. ", batchSize);

    std::lock_guard<std::mutex> lock(lock_);

    // copy data from CPU memory to GPU memory
    hipMemcpyAsync(tensor_mem_[0], &input[0],
                    batchSize * kInputPlanes * network_[0]->getH() *
                        network_[0]->getW() * sizeof(float),
                    hipMemcpyHostToDevice);

    int l = 0;
    // input
    network_[l++]->eval(batchSize, tensor_mem_[2], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // input conv

    // residual block
    for (int block = 0; block < numBlocks_; block++) {
      network_[l++]->eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                          scratch_mem_, cudnn_, cublas_);  // conv1
      network_[l++]->eval(batchSize, tensor_mem_[2], tensor_mem_[0],
                          tensor_mem_[2], scratch_mem_, cudnn_,
                          cublas_);  // conv2
    }

    // policy head
    network_[l++]->eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // pol conv
    network_[l++]->eval(batchSize, tensor_mem_[1], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // pol BN
    network_[l++]->eval(batchSize, tensor_mem_[0], tensor_mem_[1], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // pol FC
    network_[l++]->eval(batchSize, tensor_mem_[1], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_,
                        cublas_);  // pol softmax  // POLICY

    // value head
    network_[l++]->eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // value conv
    network_[l++]->eval(batchSize, tensor_mem_[2], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // value BN
    network_[l++]->eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // value FC1
    network_[l++]->eval(batchSize, tensor_mem_[2], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_,
                        cublas_);  // value FC2    // VALUE

    // copy results back to CPU memory
    hipMemcpyAsync(&op_pol[0], tensor_mem_[1],
                    policy_out_->getOutputSize(batchSize),
                    hipMemcpyDeviceToHost);
    hipError_t status = hipMemcpy(&op_val[0], tensor_mem_[2],
                                    value_out_->getOutputSize(batchSize),
                                    hipMemcpyDeviceToHost);

    if (status != hipSuccess) {
      throw Exception("Some error running cuda based eval!");
    }
  }

  ~CudnnNetwork() {
    for (auto mem : tensor_mem_) {
      if (mem) hipFree(mem);
    }
    if (scratch_mem_) hipFree(scratch_mem_);
    hipdnnDestroy(cudnn_);
    hipblasDestroy(cublas_);
  }

  std::unique_ptr<NetworkComputation> NewComputation() override {
    return std::make_unique<CudnnNetworkComputation>(this);
  }

 private:
  hipdnnHandle_t cudnn_;
  hipblasHandle_t cublas_;

  // currently only one NN eval can happen a time (we can fix this if needed by
  // allocating more memory)
  mutable std::mutex lock_;

  int numBlocks_;
  std::vector<std::unique_ptr<BaseLayer>> network_;
  BaseLayer *getLastLayer() { return network_.back().get(); }

  BaseLayer *resi_last_;
  BaseLayer *policy_out_;
  BaseLayer *value_out_;

  float *tensor_mem_[3];
  float *scratch_mem_;

  void processConvBlock(Weights::ConvBlock &block, bool foldBNLayer = false) {
    const float epsilon = 1e-5f;

    // compute reciprocal of std-dev from the variances (so that it can be just
    // multiplied)
    std::vector<float> &stddev = block.bn_stddivs;
    for (auto &&w : stddev) {
      w = 1.0f / std::sqrt(w + epsilon);
    }

    // Biases are not calculated and are typically zero but some networks might
    // still have non-zero biases.
    // Move biases to batchnorm means to make the output match without having
    // to separately add the biases.
    for (auto j = size_t{0}; j < block.bn_means.size(); j++) {
      block.bn_means[j] -= block.biases[j];
      block.biases[j] = 0.0f;
    }

    // get rid of the BN layer by adjusting weights and biases of the
    // convolution idea proposed by Henrik Forstén and first implemented in
    // leela go zero
    if (foldBNLayer) {
      const int outputs = block.biases.size();
      const int channels = block.weights.size() / (outputs * 3 * 3);

      for (auto o = 0; o < outputs; o++) {
        for (auto c = 0; c < channels; c++) {
          for (auto i = 0; i < 9; i++) {
            block.weights[o * channels * 9 + c * 9 + i] *= block.bn_stddivs[o];
          }
        }

        block.bn_means[o] *= block.bn_stddivs[o];
        block.bn_stddivs[o] = 1.0f;

        // Move means to convolution biases
        block.biases[o] = -block.bn_means[o];
        block.bn_means[o] = 0.0f;
      }
    }
  }
};

void CudnnNetworkComputation::ComputeBlocking() {
  // Convert raw_input to "expanded planes" - format the first convolutional
  // layer expects
  // TODO: can probably do this on the GPU if this becomes a bottleneck
  float *data = &(input_planes_[0][0]);
  memset(data, 0, sizeof(float) * GetBatchSize() * kInputPlanes * 8 * 8);
  auto iter = data;
  for (const auto &sample : raw_input_) {
    // CHECK_EQ(sample.size(), kInputPlanes);
    for (const auto &plane : sample) {
      for (auto bit : IterateBits(plane.mask)) {
        *(iter + bit) = plane.value;
      }
      iter += 64;
    }
  }

  network_->forwardEval(data, &(out_pol_[0][0]), &(out_val_[0]),
                        GetBatchSize());
}

}  // namespace

REGISTER_FACTORY("cudnn",
                 [](const Weights &weights, const OptionsDict &options) {
                   return std::make_unique<CudnnNetwork>(weights);
                 },
                 110);
}  // namespace lczero
