#include "hip/hip_runtime.h"
/*
  This file is part of Leela Chess Zero.
  Copyright (C) 2018 The LCZero Authors

  Leela Chess is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  Leela Chess is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with Leela Chess.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <cassert>
#include <functional>
#include <list>
#include <memory>
#include <mutex>
#include "neural/factory.h"
#include "utils/bititer.h"
#include "utils/exception.h"

#include <hipblas.h>
#include <hipDNN.h>

#define DEBUG_RAW_NPS 0

namespace lczero {
namespace {

void cudnnError(hipdnnStatus_t status, const char *file, const int &line) {
  if (status != HIPDNN_STATUS_SUCCESS) {
    char message[128];
    sprintf(message, "CUDNN error: %s (%s:%d) ", hipdnnGetErrorString(status),
            file, line);
    throw Exception(message);
  }
}

const char *cublasGetErrorString(hipblasStatus_t status) {
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "unknown cublas error";
}

void cublasError(hipblasStatus_t status, const char *file, const int &line) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    char message[128];
    sprintf(message, "CUDNN error: %s (%s:%d) ", cublasGetErrorString(status),
            file, line);
    throw Exception(message);
  }
}

void hipError_t(hipError_t status, const char *file, const int &line) {
  if (status != hipSuccess) {
    char message[128];
    sprintf(message, "CUDA error: %s (%s:%d) ", hipGetErrorString(status),
            file, line);
    throw Exception(message);
  }
}

#define reportCUDNNErrors(status) cudnnError(status, __FILE__, __LINE__)
#define reportCUBLASErrors(status) cublasError(status, __FILE__, __LINE__)
#define reportCUDAErrors(status) hipError_t(status, __FILE__, __LINE__)

// 256 MB fixed scratch memory size (hardcoded for now)
static constexpr int kCudaScratchSize = 256 * 1024 * 1024;

// hard-coded for now, no point in going above this anyway (can possibly save
// memory by reducing this)
static constexpr int kMaxBatchSize = 1024;

static constexpr int kNumOutputPolicy = 1858;

// the Layer objects only hold memory for weights, biases, etc
// memory for input and output tensors is provided by caller of Eval

class BaseLayer {
 public:
  int GetC() const { return C; }
  int GetH() const { return H; }
  int GetW() const { return W; }

  BaseLayer(int c, int h, int w, BaseLayer *ip);
  size_t GetOutputSize(int N) const { return bpe_ * N * C * H * W; }

  // input2 is optional (skip connection)
  virtual void Eval(int N, void *output, const void *input,
                    const void *input2, void *scratch, hipdnnHandle_t cudnn,
                    hipblasHandle_t cublas) = 0;

  static void enableFp16() { fp16_ = true; bpe_ = sizeof(half); }
  static bool isFp16Enabled() { return fp16_; }
 protected:
  static bool fp16_;
  static size_t bpe_;  // size of each element
  BaseLayer *input_;

  int C;  // output tensor dimensions
  int H;
  int W;
};

class ConvLayer : public BaseLayer {
 public:
  ConvLayer(BaseLayer *ip, int C, int H, int W, int size, int Cin,
            bool relu = false, bool bias = false);
  ~ConvLayer();
  void LoadWeights(float *pfilter, float *pBias , void *scratch);
  void Eval(int N, void *output, const void *input, const void *input2,
            void *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  const int c_input_;
  const int filter_size_;
  const bool use_relu_;
  const bool use_bias_;

  void *biases = nullptr;
  void *weights = nullptr;

  hipdnnFilterDescriptor_t filter_desc_;
  hipdnnConvolutionDescriptor_t conv_desc_;
  hipdnnConvolutionFwdAlgo_t convAlgo;

  hipdnnTensorDescriptor_t bias_desc_;
  hipdnnTensorDescriptor_t in_tensor_desc_;
  hipdnnTensorDescriptor_t out_tensor_desc_;
  hipdnnActivationDescriptor_t activation_;
};

class SoftMaxLayer : public BaseLayer {
 public:
  SoftMaxLayer(BaseLayer *ip);
  void Eval(int N, void *output, const void *input, const void *input2,
            void *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  hipdnnTensorDescriptor_t out_tensor_desc_;
};

class BNLayer : public BaseLayer {
 public:
  BNLayer(BaseLayer *ip, bool relu);
  ~BNLayer();

  void LoadWeights(float *cpuMeans, float *cpuVar);
  void Eval(int N, void *output, const void *input, const void *input2,
            void *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  const bool use_relu_;

  // always in float irrespective of fp16_
  // not much point in converting these to fp16
  float *means_ = nullptr;
  float *variances_ = nullptr;
};

class FCLayer : public BaseLayer {
 public:
  FCLayer(BaseLayer *ip, int C, int H, int W, bool relu, bool bias,
          bool tanh = false);
  ~FCLayer();

  void LoadWeights(float *cpuWeight, float *cpuBias, void *scratch);
  void Eval(int N, void *output, const void *input, const void *input2,
            void *scratch, hipdnnHandle_t cudnn,
            hipblasHandle_t cublas) override;

 private:
  const bool use_bias_;
  const bool use_relu_;
  const bool use_tanh_;
  void *weights_ = nullptr;
  void *biases_ = nullptr;
};

// Need memory for 3 data buffers
//  1. input for the layer
//  2. output of the layer
//  3. data from old layer for skip connection

/////////////////////////////////////////////////////////////////////////////
//                      Static variable Definations                        //
/////////////////////////////////////////////////////////////////////////////

// TODO: fp16 support
bool BaseLayer::fp16_ = false;
size_t BaseLayer::bpe_ = sizeof(float);

int divUp(int a, int b) { return (a + b - 1) / b; }

/////////////////////////////////////////////////////////////////////////////
//          Simple CUDA kernels used by certain layers                     //
/////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void addVectors_kernel(T *c, T *a, T *b, int size, int asize,
                                  int bsize, bool relu, bool useTanh) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    float aVal = 0;
    float bVal = 0;
    if (a) aVal = (float) (a[i % asize]);
    if (b) bVal = (float) (b[i % bsize]);

    float cVal = aVal + bVal;

    if (relu && (cVal < 0)) cVal = 0;

    if (useTanh) {
      // Ankan: actually it's sigmoid in leela-zero main branch??
      // see code in Network.cpp
      //    auto winrate_sig = (1.0f + std::tanh(winrate_out[0])) / 2.0f;
      // Different from lc0 branch? WHY ???
      // cVal = (1.0f + tanh(cVal)) / 2.0f;
      cVal = tanh(cVal);
    }

    c[i] = (T) cVal;
  }
}

// adds two vectors (possibly of different sizes), also do optional relu
// activation_
template <typename T>
void addVectors(T *c, T *a, T *b, int size, int asize, int bsize, bool relu,
                bool useTanh) {
  const int blockSize = 256;
  int blocks = divUp(size, blockSize);

  addVectors_kernel<<<blocks, blockSize>>>(c, a, b, size, asize, bsize, relu,
                                           useTanh);
  reportCUDAErrors(hipGetLastError());
}


__device__ half readNCHW(float *inputTensor, int n, int c, int h, int w, int Nin, int Cin, int H, int W)
{
    if (n >= Nin || c >= Cin)
        return 0;

    int index;
    index = n;
    index *= Cin;
    index += c;
    index *= H;
    index += h;
    index *= W;
    index += w;

    return (half)(inputTensor[index]);
}

__global__ void fp32NCHWtofp16NHWC_kernel(half *outputTensor, float *inputTensor, int Nin, int Cin, int Nout, int Cout, int H, int W)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= Nout * Cout * H * W)
        return;

    int index = tid;

    int c = (index % Cout);
    index /= Cout;
    int w = index % W;
    index /= W;
    int h = index % H;
    index /= H;
    int n = index;

    outputTensor[tid] = readNCHW(inputTensor, n, c, h, w, Nin, Cin, H, W);
}

void fp32NCHWtofp16NHWC(half *outputTensor, float *inputTensor, int Nin, int Cin, int Nout, int Cout, int H, int W)
{
    size_t numElements = Nout * Cout * H * W;
    const int blockSize = 256;
    int blocks = divUp(numElements, blockSize);
    fp32NCHWtofp16NHWC_kernel <<< blocks, blockSize >>> (outputTensor, inputTensor, Nin, Cin, Nout, Cout, H, W);
}


template <typename DstType, typename SrcType>
__global__ void copyTypeConverted_kernel(DstType *op, SrcType *ip, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N)
        return;

    DstType el = (DstType) ip[tid];
    op[tid] = el;
}


template <typename DstType, typename SrcType>
void copyTypeConverted(DstType *op, SrcType *ip, int N)
{
    const int blockSize = 256;
    int blocks = divUp(N, blockSize);
    copyTypeConverted_kernel <<< blocks, blockSize >>> (op, ip, N);
}

template <typename T>
__global__ void batchNormForward_kernel(T *output, const T *input,
                                        const T *skipInput, int N, int C,
                                        int H, int W, const float *means,
                                        const float *varMultipliers,
                                        bool relu) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int wIndex = 0;
  if (sizeof(T) == sizeof(float))
      wIndex = (index / (H * W)) % C;   // NCHW for fp32
  else
      wIndex = index % C;               // NHWC for fp16

  float el = input[index];
  float mean = means[wIndex];
  float varMulti = varMultipliers[wIndex];

  el -= mean;
  el *= varMulti;

  if (skipInput) el += (float) skipInput[index];

  if (relu && (el < 0)) el = 0;

  output[index] = (T) el;
}

// each thread processes single element
template <typename T>
void batchNormForward(T *output, const T *input, const T *skipInput,
                      int N, int C, int H, int W, float *means,
                      float *varMultipliers, bool relu) {
  int totalElements = N * C * H * W;
  const int blockSize = 256;
  int blocks = divUp(totalElements, blockSize);

  batchNormForward_kernel<<<blocks, blockSize>>>(
      output, input, skipInput, N, C, H, W, means, varMultipliers, relu);

  reportCUDAErrors(hipGetLastError());
}

__global__ void expandPlanes_kernel_Fp32_NCHW(float *output, const uint64_t *masks,
                                              const float *values, int n) {
  // block size of 256, same mask/val for 64 consecutive threads
  constexpr int kNumShmemElments = 256 / 64;

  __shared__ uint64_t shMasks[kNumShmemElments];
  __shared__ float shVals[kNumShmemElments];

  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int planeIndex = index >> 6;

  if (planeIndex >= n) return;

  // load inputs to shared memory
  if (threadIdx.x < kNumShmemElments) {
    shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
    shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
  }
  __syncthreads();

  uint64_t mask = shMasks[threadIdx.x >> 6];

  int sqIndex = index & 0x3F;
  float op = 0;

  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    op = shVals[threadIdx.x >> 6];
  }
  output[index] = op;
}

void expandPlanes_Fp32_NCHW(float *output, const uint64_t *masks, const float *values,
                  int n) {
  int threads = n * 8 * 8;  // each thread writes a single element
  const int blockSize = 256;
  int blocks = divUp(threads, blockSize);
  expandPlanes_kernel_Fp32_NCHW <<<blocks, blockSize>>>(output, masks, values, n);
  reportCUDAErrors(hipGetLastError());
}

// TODO: can optimize using shared memory if this becomes a bottleneck
__global__ void expandPlanes_kernel_Fp16_NHWC(half *output, const uint64_t *masks,
    const float *values, int n) {

    const int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= n * 8 * 8) return;

    const int planeIndex = index % kInputPlanes;
    const int boardIndex = index / (kInputPlanes * 8 * 8);
    const int sqIndex = (index / kInputPlanes) & 0x3F;

    uint64_t mask = masks[boardIndex * kInputPlanes + planeIndex];

    half op = 0;
    bool set = !!(mask & (1ull << sqIndex));
    if (set) {
        float val = values[boardIndex * kInputPlanes + planeIndex];
        op = (half)val;
    }
    output[index] = op;
}

void expandPlanes_Fp16_NHWC(half *output, const uint64_t *masks, const float *values,
    int n) {
    int threads = n * 8 * 8;  // each thread writes a single element
    const int blockSize = 256;
    int blocks = divUp(threads, blockSize);
    expandPlanes_kernel_Fp16_NHWC <<<blocks, blockSize >>>(output, masks, values, n);
    reportCUDAErrors(hipGetLastError());
}


BaseLayer::BaseLayer(int c, int h, int w, BaseLayer *ip)
    : C(c), H(h), W(w), input_(ip) {}

SoftMaxLayer::SoftMaxLayer(BaseLayer *ip)
    : BaseLayer(ip->GetC(), ip->GetH(), ip->GetW(), ip) {
  hipdnnCreateTensorDescriptor(&out_tensor_desc_);
}

void SoftMaxLayer::Eval(int N, void *output, const void *input,
                        const void *input2, void *scratch,
                        hipdnnHandle_t cudnn, hipblasHandle_t cublas) {
  float alpha = 1.0f, beta = 0.0f;

  // need to call this at Eval as 'N' changes :-/
  hipdnnSetTensor4dDescriptor(
      out_tensor_desc_, fp16_ ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, N, GetC(), GetH(), GetW());

  hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_ACCURATE,
                      HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha, out_tensor_desc_,
                      input, &beta, out_tensor_desc_, output);
}

ConvLayer::ConvLayer(BaseLayer *ip, int C, int H, int W, int filter, int Cin,
                     bool relu, bool bias)
    : BaseLayer(C, H, W, ip),
      filter_size_(filter),
      c_input_(Cin),
      use_relu_(relu),
      use_bias_(bias) {
  // allocate memory for weights (filter tensor) and biases
  size_t weightSize = bpe_ * Cin * C * filter_size_ * filter_size_;
  reportCUDAErrors(hipMalloc(&weights, weightSize));

  size_t biasSize = bpe_ * C;
  reportCUDAErrors(hipMalloc(&biases, biasSize));

  // create cudnn objects for various tensors, algorithms, etc
  hipdnnCreateFilterDescriptor(&filter_desc_);
  hipdnnCreateConvolutionDescriptor(&conv_desc_);
  hipdnnCreateTensorDescriptor(&out_tensor_desc_);
  hipdnnCreateTensorDescriptor(&in_tensor_desc_);
  hipdnnCreateTensorDescriptor(&bias_desc_);
  hipdnnCreateActivationDescriptor(&activation_);

  hipdnnSetFilter4dDescriptor(
      filter_desc_, fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT,
      fp16_ ? HIPDNN_TENSOR_NHWC
            : HIPDNN_TENSOR_NCHW,
      GetC(), Cin, filter_size_, filter_size_);

  reportCUDNNErrors(hipdnnSetTensor4dDescriptor(
      bias_desc_, fp16_ ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, 1, C, 1, 1));

  int padding = filter_size_ / 2;
  const bool crossCorr = 1;

  reportCUDNNErrors(hipdnnSetConvolution2dDescriptor(
      conv_desc_, padding, padding, 1, 1, 1, 1,
      crossCorr ? HIPDNN_CROSS_CORRELATION : HIPDNN_CONVOLUTION,
      fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT));

  if (fp16_)
    reportCUDNNErrors(hipdnnSetConvolutionMathType(conv_desc_, HIPDNN_TENSOR_OP_MATH));

  // TODO: dynamic selection of algorithm!
  if ((C > 32) && (!fp16_)) {
    convAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
  } else {
    convAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  }

  if (use_relu_) {
    hipdnnSetActivationDescriptor(activation_, HIPDNN_ACTIVATION_RELU,
                                 HIPDNN_NOT_PROPAGATE_NAN, 0.0);
  } else {
    hipdnnSetActivationDescriptor(activation_, HIPDNN_ACTIVATION_PATHTRU,
                                 HIPDNN_NOT_PROPAGATE_NAN, 0.0);
  }
}

void ConvLayer::LoadWeights(float *pfilter, float *pBias, void *scratch) {
  size_t weightSize = sizeof(float) * c_input_ * C * filter_size_ * filter_size_;
  size_t biasSize = sizeof(float) * C;
  if (fp16_) {
    // also need to convert from fp32 NCHW to fp16 NHWC
    // first copy from CPU memory to scratch space in GPU memory
    // and then do the type / layout conversion using a kernel
    assert(scratch);
    reportCUDAErrors(
        hipMemcpyAsync(scratch, pfilter, weightSize, hipMemcpyHostToDevice));
    fp32NCHWtofp16NHWC((half *)weights, (float*)scratch, C, c_input_, C, c_input_, filter_size_, filter_size_);

    if (pBias) {
        reportCUDAErrors(
            hipMemcpyAsync(scratch, pBias, biasSize, hipMemcpyHostToDevice));

        copyTypeConverted((half*)biases, (float *)scratch, C);
    }
  }
  else
  {
    reportCUDAErrors(
        hipMemcpyAsync(weights, pfilter, weightSize, hipMemcpyHostToDevice));

    if (pBias) {
        reportCUDAErrors(
            hipMemcpyAsync(biases, pBias, biasSize, hipMemcpyHostToDevice));
    }
    else {
        reportCUDAErrors(hipMemset(biases, biasSize, 0));
    }
  }
}

void ConvLayer::Eval(int N, void *output, const void *input,
                     const void *input2, void *scratch, hipdnnHandle_t cudnn,
                     hipblasHandle_t cublas) {
  reportCUDNNErrors(hipdnnSetTensor4dDescriptor(
      out_tensor_desc_, fp16_ ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, N, C, H, W));

  reportCUDNNErrors(hipdnnSetTensor4dDescriptor(
      in_tensor_desc_, fp16_ ? HIPDNN_TENSOR_NHWC : HIPDNN_TENSOR_NCHW,
      fp16_ ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT, N, c_input_, H, W));

  float alpha = 1.0f, beta = 0.0f;

  if (!(use_relu_ || use_bias_)) {
    reportCUDNNErrors(hipdnnConvolutionForward(
        cudnn, &alpha, in_tensor_desc_, input, filter_desc_, weights,
        conv_desc_, convAlgo, scratch, kCudaScratchSize, &beta,
        out_tensor_desc_, output));
  } else if (input2) {
    // fused bias + sum + relu!
    reportCUDNNErrors(cudnnConvolutionBiasActivationForward(
        cudnn, &alpha, in_tensor_desc_, input, filter_desc_, weights,
        conv_desc_, convAlgo, scratch, kCudaScratchSize, &alpha,
        out_tensor_desc_, input2, bias_desc_, biases, activation_,
        out_tensor_desc_, output));
  } else {
    reportCUDNNErrors(cudnnConvolutionBiasActivationForward(
        cudnn, &alpha, in_tensor_desc_, input, filter_desc_, weights,
        conv_desc_, convAlgo, scratch, kCudaScratchSize, &beta,
        out_tensor_desc_, output, bias_desc_, biases, activation_,
        out_tensor_desc_, output));
  }
}

ConvLayer::~ConvLayer() {
  reportCUDAErrors(hipFree(weights));
  reportCUDAErrors(hipFree(biases));
}

BNLayer::BNLayer(BaseLayer *ip, bool relu)
    : BaseLayer(ip->GetC(), ip->GetH(), ip->GetW(), ip), use_relu_(relu) {
  size_t weightSize = sizeof(float) * C;

  reportCUDAErrors(hipMalloc(&means_, weightSize));
  reportCUDAErrors(hipMalloc(&variances_, weightSize));
}

void BNLayer::LoadWeights(float *cpuMeans, float *cpuVar) {
  size_t weightSize = sizeof(float) * C;
  reportCUDAErrors(
      hipMemcpyAsync(means_, cpuMeans, weightSize, hipMemcpyHostToDevice));
  reportCUDAErrors(
      hipMemcpyAsync(variances_, cpuVar, weightSize, hipMemcpyHostToDevice));
}

void BNLayer::Eval(int N, void *output, const void *input,
                   const void *input2, void *scratch, hipdnnHandle_t cudnn,
                   hipblasHandle_t cublas) {
  if (fp16_)
    batchNormForward((half*)output, (const half*)input, (const half*) input2, 
                     N, C, H, W, means_, variances_, use_relu_);
  else
    batchNormForward((float*)output, (const float*)input, (const float*) input2,
                     N, C, H, W, means_, variances_, use_relu_);
}

BNLayer::~BNLayer() {
  reportCUDAErrors(hipFree(means_));
  reportCUDAErrors(hipFree(variances_));
}

FCLayer::FCLayer(BaseLayer *ip, int C, int H, int W, bool relu, bool bias,
                 bool tanh)
    : BaseLayer(C, H, W, ip),
      use_relu_(relu),
      use_bias_(bias),
      use_tanh_(tanh) {
  size_t weightSize = bpe_ * C * H * W * ip->GetC() * ip->GetH() * ip->GetW();
  size_t biasSize = bpe_ * C * H * W;
  reportCUDAErrors(hipMalloc(&weights_, weightSize));
  if (use_bias_) {
    reportCUDAErrors(hipMalloc(&biases_, biasSize));
  } else {
    biases_ = nullptr;
  }
}

void FCLayer::LoadWeights(float *cpuWeight, float *cpuBias, void *scratch) {
  size_t numWeights = C * H * W * input_->GetC() * input_->GetH() * input_->GetW();
  size_t weightSize = sizeof(float) * numWeights;
  size_t numBiases = C * H * W;
  size_t biasSize = sizeof(float) * numBiases;

  if (fp16_) {
    // also need to convert from fp32 to fp16 
    assert(scratch);
    reportCUDAErrors(
        hipMemcpyAsync(scratch, cpuWeight, weightSize, hipMemcpyHostToDevice));

    //copyTypeConverted((half*)weights_, (float *)scratch, numWeights);
    fp32NCHWtofp16NHWC((half *)weights_, (float*)scratch, numBiases, input_->GetC(), numBiases, input_->GetC(), input_->GetH(), input_->GetW());

    if (cpuBias) {
        reportCUDAErrors(
            hipMemcpyAsync(scratch, cpuBias, biasSize, hipMemcpyHostToDevice));
        copyTypeConverted((half*)biases_, (float *)scratch, numBiases);
    }
  }
  else
  {
    reportCUDAErrors(
        hipMemcpyAsync(weights_, cpuWeight, weightSize, hipMemcpyHostToDevice));
    if (use_bias_) {
        reportCUDAErrors(
            hipMemcpyAsync(biases_, cpuBias, biasSize, hipMemcpyHostToDevice));
    }
  }
}

// taken from: https://devtalk.nvidia.com/default/topic/883897/error-when-trying-to-use-half-fp16-/
/*
Copyright (c) 2015, Norbert Juffa
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

half uint16_as_fp16(uint16_t a)
{
    half res;
#if defined (__cplusplus)
    memcpy(&res, &a, sizeof(res));
#else /* __cplusplus */
    volatile union {
        __fp16 f;
        uint16_t i;
    } cvt;
    cvt.i = a;
    res = cvt.f;
#endif /* __cplusplus */
    return res;
}

uint32_t fp32_as_uint32(float a)
{
    uint32_t res;
#if defined (__cplusplus)
    memcpy(&res, &a, sizeof(res));
#else /* __cplusplus */
    volatile union {
        float f;
        uint32_t i;
    } cvt;
    cvt.f = a;
    res = cvt.i;
#endif /* __cplusplus */
    return res;
}

/* host version of device function __float2half_rn() */
half float2half_rn(float a)
{
    uint32_t ia = fp32_as_uint32(a);
    uint16_t ir;

    ir = (ia >> 16) & 0x8000;
    if ((ia & 0x7f800000) == 0x7f800000) {
        if ((ia & 0x7fffffff) == 0x7f800000) {
            ir |= 0x7c00; /* infinity */
        }
        else {
            ir = 0x7fff; /* canonical NaN */
        }
    }
    else if ((ia & 0x7f800000) >= 0x33000000) {
        int shift = (int)((ia >> 23) & 0xff) - 127;
        if (shift > 15) {
            ir |= 0x7c00; /* infinity */
        }
        else {
            ia = (ia & 0x007fffff) | 0x00800000; /* extract mantissa */
            if (shift < -14) { /* denormal */
                ir |= ia >> (-1 - shift);
                ia = ia << (32 - (-1 - shift));
            }
            else { /* normal */
                ir |= ia >> (24 - 11);
                ia = ia << (32 - (24 - 11));
                ir = ir + ((14 + shift) << 10);
            }
            /* IEEE-754 round to nearest of even */
            if ((ia > 0x80000000) || ((ia == 0x80000000) && (ir & 1))) {
                ir++;
            }
        }
    }
    return uint16_as_fp16(ir);
}

void FCLayer::Eval(int N, void *outputTensor, const void *inputTensor,
                   const void *input2, void *scratch, hipdnnHandle_t cudnn,
                   hipblasHandle_t cublas) {
  int numOutputs = C * H * W;
  int numInputs = input_->GetC() * input_->GetH() * input_->GetW();

  if (fp16_) {
    half alpha = float2half_rn(1.0f), beta = float2half_rn(0.0f);
    reportCUBLASErrors(hipblasHgemm(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                       numOutputs, N, numInputs,
                       &alpha,
                       (half *)weights_, numInputs,
                       (half *)inputTensor, numInputs,
                       &beta,
                       (half *)outputTensor, numOutputs));

    if (use_bias_ || use_relu_ || use_tanh_) {
        addVectors((half*)outputTensor, (half*)biases_, (half*)outputTensor, numOutputs * N,
            numOutputs, numOutputs * N, use_relu_, use_tanh_);
    }

  } else {
    float alpha = 1.0f, beta = 0.0f;
    reportCUBLASErrors(hipblasSgemm(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, numOutputs,
                                   N, numInputs, &alpha, (float*)weights_, numInputs,
                                   (float*)inputTensor, numInputs, &beta, (float*)outputTensor,
                                   numOutputs));

    if (use_bias_ || use_relu_ || use_tanh_) {
      addVectors((float*)outputTensor, (float *)biases_, (float*)outputTensor, numOutputs * N,
                 numOutputs, numOutputs * N, use_relu_, use_tanh_);
    }
  }
}

FCLayer::~FCLayer() {
  reportCUDAErrors(hipFree(weights_));
  reportCUDAErrors(hipFree(biases_));
}

struct InputsOutputs {
  InputsOutputs() {
    reportCUDAErrors(hipHostAlloc(
        &input_masks_mem_, kMaxBatchSize * kInputPlanes * sizeof(uint64_t),
        hipHostMallocMapped));
    reportCUDAErrors(
        hipHostGetDevicePointer(&input_masks_mem_gpu_, input_masks_mem_, 0));

    reportCUDAErrors(hipHostAlloc(&input_val_mem_,
                                   kMaxBatchSize * kInputPlanes * sizeof(float),
                                   hipHostMallocMapped));
    reportCUDAErrors(
        hipHostGetDevicePointer(&input_val_mem_gpu_, input_val_mem_, 0));

    reportCUDAErrors(hipHostAlloc(
        &op_policy_mem_, kMaxBatchSize * kNumOutputPolicy * sizeof(float),
        hipHostMallocMapped));
    reportCUDAErrors(
        hipHostGetDevicePointer(&op_policy_mem_gpu_, op_policy_mem_, 0));

    reportCUDAErrors(hipHostAlloc(
        &op_value_mem_, kMaxBatchSize * sizeof(float), hipHostMallocMapped));
    reportCUDAErrors(
        hipHostGetDevicePointer(&op_value_mem_gpu_, op_value_mem_, 0));
  }
  ~InputsOutputs() {
    reportCUDAErrors(hipHostFree(input_masks_mem_));
    reportCUDAErrors(hipHostFree(input_val_mem_));
    reportCUDAErrors(hipHostFree(op_policy_mem_));
    reportCUDAErrors(hipHostFree(op_value_mem_));
  }
  uint64_t *input_masks_mem_;
  float *input_val_mem_;
  float *op_policy_mem_;
  float *op_value_mem_;

  // GPU pointers for the above allocations
  uint64_t *input_masks_mem_gpu_;
  float *input_val_mem_gpu_;
  float *op_policy_mem_gpu_;
  float *op_value_mem_gpu_;
};

// This namespace should be closed at the very end of file, but otherwise
// there are nvcc warnings. Weird way to silence warnings.
}  // namespace

class CudnnNetwork;
class CudnnNetworkComputation : public NetworkComputation {
 public:
  CudnnNetworkComputation(CudnnNetwork *network);
  ~CudnnNetworkComputation();

  void AddInput(InputPlanes &&input) override {
    auto iterMask =
        &inputs_outputs_->input_masks_mem_[batch_size_ * kInputPlanes];
    auto iterVal = &inputs_outputs_->input_val_mem_[batch_size_ * kInputPlanes];

    int i = 0;
    for (const auto &plane : input) {
      iterMask[i] = plane.mask;
      iterVal[i] = plane.value;
      i++;
    }

    batch_size_++;
  }

  void ComputeBlocking() override;

  int GetBatchSize() const override { return batch_size_; }

  float GetQVal(int sample) const override {
    return inputs_outputs_->op_value_mem_[sample];
  }
  float GetPVal(int sample, int move_id) const override {
    return inputs_outputs_->op_policy_mem_[sample * kNumOutputPolicy + move_id];
  }

 private:
  // memory holding inputs, outputs
  std::unique_ptr<InputsOutputs> inputs_outputs_;
  int batch_size_;

  CudnnNetwork *network_;
};

class CudnnNetwork : public Network {
 public:
  CudnnNetwork(Weights weights, const OptionsDict &options) {
    gpuId_ = options.GetOrDefault<int>("gpu", 0);
    int tryFp16 = options.GetOrDefault<int>("fp16", 0);

    int totalGPUs;
    reportCUDAErrors(hipGetDeviceCount(&totalGPUs));

    if (gpuId_ >= totalGPUs)
      throw Exception("Invalid GPU Id: " + std::to_string(gpuId_));

    // select GPU to run on (for *the current* thread)
    reportCUDAErrors(hipSetDevice(gpuId_));

    reportCUDNNErrors(hipdnnCreate(&cudnn_));
    reportCUBLASErrors(hipblasCreate(&cublas_));

    if (tryFp16) {
        // check if the GPU support fp16 (Volta+)
        // enable fp16 only if all devices on which we are trying to run 
        // have fp16 support (TODO: can fix this limitation if needed)
        hipDeviceProp_t deviceProp = {};
        hipGetDeviceProperties(&deviceProp, gpuId_);
        if (deviceProp.major >= 7) {
            BaseLayer::enableFp16();
            reportCUBLASErrors(hipblasSetMathMode(cublas_, HIPBLAS_TENSOR_OP_MATH));
        } else {
            throw Exception("Your GPU doesn't support FP16");
        }
    } else {
        if (BaseLayer::isFp16Enabled()) {
            throw Exception("Different fp16 setting for different GPUs not yet supported");
        }
    }

    const int numInputPlanes = kInputPlanes;
    const int numFilters = weights.input.biases.size();

    numBlocks_ = weights.residual.size();

    // 0. process weights
    processConvBlock(weights.input, true);
    for (auto i = size_t{0}; i < numBlocks_; i++) {
      processConvBlock(weights.residual[i].conv1, true);
      processConvBlock(weights.residual[i].conv2, true);
    }
    processConvBlock(weights.policy);
    processConvBlock(weights.value);

    // 1. allocate scratch space (used internally by cudnn to run convolutions,
    //     and also for format/layout conversion for weights)
    reportCUDAErrors(hipMalloc(&scratch_mem_, kCudaScratchSize));


    // 2. build the network, and copy the weights to GPU memory
    // input
    {
      auto inputConv = std::make_unique<ConvLayer>(nullptr, numFilters, 8, 8, 3,
                                                   numInputPlanes, true, true);
      inputConv->LoadWeights(&weights.input.weights[0],
                             &weights.input.biases[0], 
                             scratch_mem_);
      network_.emplace_back(std::move(inputConv));
    }

    // residual block
    for (int block = 0; block < weights.residual.size(); block++) {
      auto conv1 = std::make_unique<ConvLayer>(getLastLayer(), numFilters, 8, 8,
                                               3, numFilters, true, true);
      conv1->LoadWeights(&weights.residual[block].conv1.weights[0],
                         &weights.residual[block].conv1.biases[0],
                         scratch_mem_);
      network_.emplace_back(std::move(conv1));

      auto conv2 = std::make_unique<ConvLayer>(getLastLayer(), numFilters, 8, 8,
                                               3, numFilters, true, true);
      conv2->LoadWeights(&weights.residual[block].conv2.weights[0],
                         &weights.residual[block].conv2.biases[0],
                         scratch_mem_);
      network_.emplace_back(std::move(conv2));
    }

    resi_last_ = getLastLayer();

    // policy head
    {
      auto convPol = std::make_unique<ConvLayer>(
          resi_last_, weights.policy.bn_means.size(), 8, 8, 1, numFilters);
      convPol->LoadWeights(&weights.policy.weights[0], nullptr, scratch_mem_);
      network_.emplace_back(std::move(convPol));

      auto BNPol = std::make_unique<BNLayer>(getLastLayer(), true);
      BNPol->LoadWeights(&weights.policy.bn_means[0],
                         &weights.policy.bn_stddivs[0]);
      network_.emplace_back(std::move(BNPol));

      auto FCPol = std::make_unique<FCLayer>(
          getLastLayer(), weights.ip_pol_b.size(), 1, 1, false, true);
      FCPol->LoadWeights(&weights.ip_pol_w[0], &weights.ip_pol_b[0], scratch_mem_);
      network_.emplace_back(std::move(FCPol));

      auto softmaxPol = std::make_unique<SoftMaxLayer>(getLastLayer());
      network_.emplace_back(std::move(softmaxPol));
    }
    policy_out_ = getLastLayer();

    // Value head
    {
      auto convVal = std::make_unique<ConvLayer>(
          resi_last_, weights.value.bn_means.size(), 8, 8, 1, numFilters);
      convVal->LoadWeights(&weights.value.weights[0], nullptr, scratch_mem_);
      network_.emplace_back(std::move(convVal));

      auto BNVal = std::make_unique<BNLayer>(getLastLayer(), true);
      BNVal->LoadWeights(&weights.value.bn_means[0],
                         &weights.value.bn_stddivs[0]);
      network_.emplace_back(std::move(BNVal));

      auto FCVal1 = std::make_unique<FCLayer>(
          getLastLayer(), weights.ip1_val_b.size(), 1, 1, true, true);
      FCVal1->LoadWeights(&weights.ip1_val_w[0], &weights.ip1_val_b[0], scratch_mem_);
      network_.emplace_back(std::move(FCVal1));

      auto FCVal2 =
          std::make_unique<FCLayer>(getLastLayer(), 1, 1, 1, false, true, true);
      FCVal2->LoadWeights(&weights.ip2_val_w[0], &weights.ip2_val_b[0], scratch_mem_);
      network_.emplace_back(std::move(FCVal2));
    }
    value_out_ = getLastLayer();

    // 3. allocate GPU memory for running the network
    //    - three buffers of max size are enough (one to hold input, second to
    //    hold output and third to hold skip connection's input)
    size_t maxSize = resi_last_->GetOutputSize(kMaxBatchSize);
    for (auto &mem : tensor_mem_) {
      reportCUDAErrors(hipMalloc(&mem, maxSize));
      reportCUDAErrors(hipMemset(mem, 0, maxSize));
    }

    // printf("Allocated %d bytes of GPU memory to run the network\n", 3 *
    // maxSize);
  }

  void forwardEval(InputsOutputs *io, int batchSize) {
    std::lock_guard<std::mutex> lock(lock_);

#if DEBUG_RAW_NPS == 1
    auto t_start = std::chrono::high_resolution_clock::now();
#endif

    // expand packed planes to full planes
    uint64_t *ipDataMasks = io->input_masks_mem_gpu_;
    float *ipDataValues = io->input_val_mem_gpu_;

    if (BaseLayer::isFp16Enabled()) {
        expandPlanes_Fp16_NHWC((half*)(tensor_mem_[0]), ipDataMasks, ipDataValues,
            batchSize * kInputPlanes);
    } else {
        expandPlanes_Fp32_NCHW((float*)(tensor_mem_[0]), ipDataMasks, ipDataValues,
            batchSize * kInputPlanes);
    }

    float *opPol = io->op_policy_mem_gpu_;
    float *opVal = io->op_value_mem_gpu_;

    int l = 0;
    // input
    network_[l++]->Eval(batchSize, tensor_mem_[2], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // input conv

    // residual block
    for (int block = 0; block < numBlocks_; block++) {
      network_[l++]->Eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                          scratch_mem_, cudnn_, cublas_);  // conv1

      network_[l++]->Eval(batchSize, tensor_mem_[2], tensor_mem_[0],
                          tensor_mem_[2], scratch_mem_, cudnn_,
                          cublas_);  // conv2
    }

    // policy head
    network_[l++]->Eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // pol conv
    network_[l++]->Eval(batchSize, tensor_mem_[1], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // pol BN
    network_[l++]->Eval(batchSize, tensor_mem_[0], tensor_mem_[1], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // pol FC
    if (BaseLayer::isFp16Enabled()) {
      // TODO: consider softmax layer that writes directly to fp32
      network_[l++]->Eval(batchSize, tensor_mem_[1], tensor_mem_[0], nullptr, scratch_mem_,
                          cudnn_,
                          cublas_);  // pol softmax
      copyTypeConverted(opPol, (half *)(tensor_mem_[1]), batchSize * kNumOutputPolicy); // POLICY
    } else {
      network_[l++]->Eval(batchSize, opPol, tensor_mem_[0], nullptr, scratch_mem_,
                          cudnn_,
                          cublas_);  // pol softmax  // POLICY
    }

    // value head
    network_[l++]->Eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // value conv
    network_[l++]->Eval(batchSize, tensor_mem_[2], tensor_mem_[0], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // value BN
    network_[l++]->Eval(batchSize, tensor_mem_[0], tensor_mem_[2], nullptr,
                        scratch_mem_, cudnn_, cublas_);  // value FC1
    if (BaseLayer::isFp16Enabled()) {
      // TODO: consider fusing the bias-add of FC2 with format conversion
      network_[l++]->Eval(batchSize, tensor_mem_[2], tensor_mem_[0], nullptr, scratch_mem_,
                          cudnn_,
                          cublas_);  // value FC2
      copyTypeConverted(opVal, (half *)(tensor_mem_[2]), batchSize); // VALUE
    } else {
      network_[l++]->Eval(batchSize, opVal, tensor_mem_[0], nullptr, scratch_mem_,
                          cudnn_,
                          cublas_);  // value FC2    // VALUE
    }
    reportCUDAErrors(hipDeviceSynchronize());

#if DEBUG_RAW_NPS == 1
    const int reportingCalls = 100;
    static int numCalls = 0;
    static int sumBatchSize = 0;
    static double totalTime = 0;

    sumBatchSize += batchSize;
    numCalls++;

    auto t_end = std::chrono::high_resolution_clock::now();

    double dt = std::chrono::duration<double>(t_end - t_start).count();
    totalTime += dt;
    if (numCalls == reportingCalls) {
      double avgBatchSize = ((double)sumBatchSize) / numCalls;
      printf("\nAvg batch size: %lf, NN eval time: %lf seconds per %d evals\n",
             avgBatchSize, totalTime, sumBatchSize);
      sumBatchSize = 0;
      totalTime = 0;
      numCalls = 0;
    }
#endif
  }

  ~CudnnNetwork() {
    for (auto mem : tensor_mem_) {
      if (mem) reportCUDAErrors(hipFree(mem));
    }
    if (scratch_mem_) reportCUDAErrors(hipFree(scratch_mem_));
    hipdnnDestroy(cudnn_);
    hipblasDestroy(cublas_);
  }

  std::unique_ptr<NetworkComputation> NewComputation() override {
    // set correct gpu id for this computation (as it might have been called
    // from a different thread)
    reportCUDAErrors(hipSetDevice(gpuId_));
    return std::make_unique<CudnnNetworkComputation>(this);
  }

  std::unique_ptr<InputsOutputs> GetInputsOutputs() {
    std::lock_guard<std::mutex> lock(inputs_outputs_lock_);
    if (free_inputs_outputs_.empty()) {
      return std::make_unique<InputsOutputs>();
    } else {
      std::unique_ptr<InputsOutputs> resource =
          std::move(free_inputs_outputs_.front());
      free_inputs_outputs_.pop_front();
      return resource;
    }
  }

  void ReleaseInputsOutputs(std::unique_ptr<InputsOutputs> resource) {
    std::lock_guard<std::mutex> lock(inputs_outputs_lock_);
    free_inputs_outputs_.push_back(std::move(resource));
  }

  // Apparently nvcc doesn't see constructor invocations through make_unique.
  // This function invokes constructor just to please complier and silence
  // warning. Is never called (but compiler thinks that it could).
  void UglyFunctionToSilenceNvccWarning() { InputsOutputs io; }

 private:
  hipdnnHandle_t cudnn_;
  hipblasHandle_t cublas_;
  int gpuId_;

  // currently only one NN Eval can happen a time (we can fix this if needed by
  // allocating more memory)
  mutable std::mutex lock_;

  int numBlocks_;
  std::vector<std::unique_ptr<BaseLayer>> network_;
  BaseLayer *getLastLayer() { return network_.back().get(); }

  BaseLayer *resi_last_;
  BaseLayer *policy_out_;
  BaseLayer *value_out_;

  void *tensor_mem_[3];
  void *scratch_mem_;

  mutable std::mutex inputs_outputs_lock_;
  std::list<std::unique_ptr<InputsOutputs>> free_inputs_outputs_;

  void processConvBlock(Weights::ConvBlock &block, bool foldBNLayer = false) {
    const float epsilon = 1e-5f;

    // compute reciprocal of std-dev from the variances (so that it can be just
    // multiplied)
    std::vector<float> &stddev = block.bn_stddivs;
    for (auto &&w : stddev) {
      w = 1.0f / std::sqrt(w + epsilon);
    }

    // Biases are not calculated and are typically zero but some networks might
    // still have non-zero biases.
    // Move biases to batchnorm means to make the output match without having
    // to separately add the biases.
    for (auto j = size_t{0}; j < block.bn_means.size(); j++) {
      block.bn_means[j] -= block.biases[j];
      block.biases[j] = 0.0f;
    }

    // get rid of the BN layer by adjusting weights and biases of the
    // convolution idea proposed by Henrik Forst�n and first implemented in
    // leela go zero
    if (foldBNLayer) {
      const int outputs = block.biases.size();
      const int channels = block.weights.size() / (outputs * 3 * 3);

      for (auto o = 0; o < outputs; o++) {
        for (auto c = 0; c < channels; c++) {
          for (auto i = 0; i < 9; i++) {
            block.weights[o * channels * 9 + c * 9 + i] *= block.bn_stddivs[o];
          }
        }

        block.bn_means[o] *= block.bn_stddivs[o];
        block.bn_stddivs[o] = 1.0f;

        // Move means to convolution biases
        block.biases[o] = -block.bn_means[o];
        block.bn_means[o] = 0.0f;
      }
    }
  }
};

CudnnNetworkComputation::CudnnNetworkComputation(CudnnNetwork *network)
    : network_(network) {
  batch_size_ = 0;
  inputs_outputs_ = network_->GetInputsOutputs();
}

CudnnNetworkComputation::~CudnnNetworkComputation() {
  network_->ReleaseInputsOutputs(std::move(inputs_outputs_));
}

void CudnnNetworkComputation::ComputeBlocking() {
  network_->forwardEval(inputs_outputs_.get(), GetBatchSize());
}

REGISTER_NETWORK("cudnn", CudnnNetwork, 110)

}  // namespace lczero
